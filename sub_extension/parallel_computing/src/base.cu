#include "hip/hip_runtime.h"
#include "base.cuh"
#include <iostream>

__global__ void print_one()
{
    printf("1\n");
}

void hello()
{
    std::cout << "Hello World from GPU!\n";
    print_one<<<1, 2>>>();
}