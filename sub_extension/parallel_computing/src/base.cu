#include "hip/hip_runtime.h"
#include "base.h"
#include <iostream>
#include <stdio.h>

__global__ void print_one()
{
    printf("1\n");
}

void hello()
{
    std::cout << "Hello World from GPU!\n";
    print_one<<<1, 2>>>();
}